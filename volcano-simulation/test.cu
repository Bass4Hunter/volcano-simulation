
#include <hip/hip_runtime.h>
/*

    EJECUTAR SI SE DESEA PROBRAR EL FUNCIONAMIENTO DE CUDA

*/

/*
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <cuda.h>
#include <cuda_runtime.h>
#include "device_launch_parameters.h"
#include <time.h>

using namespace std;
// Kernel function to add the elements of two arrays
__global__

void add(int n, float* x, float* y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    long N = 1 << 20;
    float* x, * y;

    // Allocate Unified Memory � accessible from CPU or GPU
    cudaMallocManaged(&x, N * sizeof(float));
    cudaMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (long i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    unsigned int time0, time1;
    time0 = clock();

    add << < 1, 1 >> > (N, x, y);

    // Wait for GPU to finish before accessing on host
    cudaDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
        //std::cout << "y[i] = " << y[i] << endl;
    }

    std::cout << "Max error: " << maxError << std::endl;
    time1 = clock();
    double timeCreation = double(time1 - time0) / CLOCKS_PER_SEC;
    printf("time %lf", timeCreation);
    // Free memory
    cudaFree(x);
    cudaFree(y);

    return 0;
}*/